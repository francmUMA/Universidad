#include "hip/hip_runtime.h"
#include <stdio.h>

#define N 2048 * 2048 // Number of elements in each vector

/*
 * Optimize this already-accelerated codebase. Work iteratively,
 * and use nsys to support your work.
 *
 * Aim to profile `saxpy` (without modifying `N`) running under
 * 20us.
 *
 * Some bugs have been placed in this codebase for your edification.
 */

__global__ void saxpy(int * a, int * b, int * c)
{
    
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if ( tid < N )
        c[tid] = 2 * a[tid] + b[tid];
}

__global__ void initialize(int * a, int * b, int * c){
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if ( tid < N ){
        a[tid] = 2;
        b[tid] = 1;
        c[tid] = 0;
    }
}

int main()
{
    int *a, *b, *c;
    int deviceId;
    hipGetDevice(&deviceId);

    int size = N * sizeof (int); // The total number of bytes per vector

    hipMallocManaged(&a, size);
    hipMallocManaged(&b, size);
    hipMallocManaged(&c, size);

    int threads_per_block = 128;
    int number_of_blocks = (N / threads_per_block) + 1;
    
    hipMemPrefetchAsync(a, size, deviceId);
    hipMemPrefetchAsync(b, size, deviceId);
    hipMemPrefetchAsync(c, size, deviceId);
    
    // Initialize memory
    initialize <<< number_of_blocks, threads_per_block >>> ( a, b, c );
    
    //Run main kernel
    saxpy <<< number_of_blocks, threads_per_block >>> ( a, b, c );
    hipMemPrefetchAsync(c, size, deviceId);
    hipDeviceSynchronize();

    // Print out the first and last 5 values of c for a quality check
    for( int i = 0; i < 5; ++i )
        printf("c[%d] = %d, ", i, c[i]);
    printf ("\n");
    for( int i = N-5; i < N; ++i )
        printf("c[%d] = %d, ", i, c[i]);
    printf ("\n");

    hipFree( a ); hipFree( b ); hipFree( c );
}
