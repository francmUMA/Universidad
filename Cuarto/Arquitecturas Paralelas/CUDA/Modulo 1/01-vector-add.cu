
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <assert.h>

void initWith(float num, float *a, int N)
{
  for(int i = 0; i < N; ++i)
  {
    a[i] = num;
  }
}

__global__
void addVectorsInto(float *result, float *a, float *b, int N)
{
  int index = threadIdx.x + blockIdx.x * blockDim.x;
  result[index] = a[index] + b[index];
}

void checkElementsAre(float target, float *array, int N)
{
  for(int i = 0; i < N; i++)
  {
    if(array[i] != target)
    {
      printf("FAIL: array[%d] - %0.0f does not equal %0.0f\n", i, array[i], target);
      exit(1);
    }
  }
  printf("SUCCESS! All values added correctly.\n");
}

int main()
{
  const int N = 2<<20;
  size_t size = N * sizeof(float);

  float *a;
  float *b;
  float *c;

  hipMallocManaged(&a, size);
  hipMallocManaged(&b, size);
  hipMallocManaged(&c, size);

  initWith(3, a, N);
  initWith(4, b, N);
  initWith(0, c, N);

  size_t threadsPerBlock;
  size_t numberOfBlocks;

  threadsPerBlock = 256;
  numberOfBlocks = N/256;

  addVectorsInto<<<numberOfBlocks, threadsPerBlock>>>(c, a, b, N);
  
  hipDeviceSynchronize();

  checkElementsAre(7, c, N);
    
  hipFree(a);
  hipFree(b);
  hipFree(c);
}
